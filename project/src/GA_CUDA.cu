#include "hip/hip_runtime.h"
#include "GA_CUDA.h"
#include "GA_CPU.h"  // Refer to the CPU version for synchronization functions
#include "TSP.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <iostream>
#include <chrono>

namespace GA {

using namespace std::chrono;

// ---------------------------------------------------------------------
// Kernel: Compute fitness for each individual
// ---------------------------------------------------------------------
__global__ void computeFitnessKernel(const int *d_population,
                                       const float *d_distanceMatrix,
                                       float *d_fitness,
                                       int numCities,
                                       int popCount)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < popCount) {
        float totalDist = 0.0f;
        int base = idx * numCities;
        for (int i = 0; i < numCities - 1; i++) {
            int c1 = d_population[base + i];
            int c2 = d_population[base + i + 1];
            totalDist += d_distanceMatrix[c1 * numCities + c2];
        }
        int lastCity = d_population[base + numCities - 1];
        int firstCity = d_population[base];
        totalDist += d_distanceMatrix[lastCity * numCities + firstCity];
        d_fitness[idx] = (totalDist <= 0.0f) ? 0.0f : (1.0f / totalDist);
    }
}

// ---------------------------------------------------------------------
// Kernel: Order Crossover (OX) for a pair of parents.
// ---------------------------------------------------------------------
__global__ void crossoverKernel(const int *d_parentA, const int *d_parentB,
                                  int *d_child1, int *d_child2,
                                  int numPairs, int numCities,
                                  float crossoverProb, unsigned long seed)
{
    int pairIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pairIdx >= numPairs) return;

    // Initialize CURAND state with given seed and thread-specific offset
    hiprandState state;
    hiprand_init(seed, pairIdx, 0, &state);

    int base = pairIdx * numCities;
    int *child1 = d_child1 + base;
    int *child2 = d_child2 + base;

    float r = hiprand_uniform(&state);
    // If crossover is not applied, copy parents directly to children
    if (r >= crossoverProb) {
        for (int i = 0; i < numCities; i++) {
            child1[i] = d_parentA[base + i];
            child2[i] = d_parentB[base + i];
        }
        return;
    }

    // Check if parents are identical; if yes, copy them directly
    bool identical = true;
    for (int i = 0; i < numCities; i++) {
        if (d_parentA[base + i] != d_parentB[base + i]) {
            identical = false;
            break;
        }
    }
    if (identical) {
        for (int i = 0; i < numCities; i++) {
            child1[i] = d_parentA[base + i];
            child2[i] = d_parentB[base + i];
        }
        return;
    }

    // Initialize children with -1 values
    for (int i = 0; i < numCities; i++) {
        child1[i] = -1;
        child2[i] = -1;
    }
    int p1 = hiprand(&state) % numCities;
    int p2 = hiprand(&state) % numCities;
    if (p1 > p2) { int tmp = p1; p1 = p2; p2 = tmp; }
    // Copy segment from parents to children
    for (int i = p1; i <= p2; i++) {
        child1[i] = d_parentA[base + i];
        child2[i] = d_parentB[base + i];
    }
    // Fill child1 with genes from parentB
    int idx = (p2 + 1) % numCities;
    for (int i = 0; i < numCities; i++) {
        int pos = (p2 + 1 + i) % numCities;
        int gene = d_parentB[base + pos];
        bool found = false;
        for (int j = p1; j <= p2; j++) {
            if (child1[j] == gene) { found = true; break; }
        }
        if (!found) {
            child1[idx] = gene;
            idx = (idx + 1) % numCities;
        }
    }
    // Fill child2 with genes from parentA
    idx = (p2 + 1) % numCities;
    for (int i = 0; i < numCities; i++) {
        int pos = (p2 + 1 + i) % numCities;
        int gene = d_parentA[base + pos];
        bool found = false;
        for (int j = p1; j <= p2; j++) {
            if (child2[j] == gene) { found = true; break; }
        }
        if (!found) {
            child2[idx] = gene;
            idx = (idx + 1) % numCities;
        }
    }
}

// ---------------------------------------------------------------------
// Kernel: Mutation: Each thread processes one individual
// ---------------------------------------------------------------------
__global__ void mutationKernel(int *d_offspring, int totalIndividuals, int numCities,
                               float mutationProb, unsigned long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalIndividuals) return;

    // Initialize CURAND state for this thread
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    int start = idx * numCities;
    for (int i = 0; i < numCities; i++) {
        float r = hiprand_uniform(&state);
        if (r < mutationProb) {
            int j = hiprand(&state) % numCities;
            int tmp = d_offspring[start + i];
            d_offspring[start + i] = d_offspring[start + j];
            d_offspring[start + j] = tmp;
        }
    }
}

// ---------------------------------------------------------------------
// Kernel: Replacement
// Each thread processes one parent pair (with two children).
// It selects the two best individuals from {parent A, parent B, child1, child2}
// and writes them back to the flattened population.
// ---------------------------------------------------------------------
__global__ void replacementKernel(
    int *d_population, float *d_populationFitness,
    const int *d_parentA, const int *d_parentB,
    const float *d_parentFitness,
    const int *d_child1, const int *d_child2,
    const float *d_childFitness,
    int numPairs, int numCities, int totalPairs)
{
    int pairIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pairIdx >= numPairs) return;

    int base = pairIdx * numCities;
    float fits[4] = {
        d_parentFitness[2 * pairIdx],
        d_parentFitness[2 * pairIdx + 1],
        d_childFitness[pairIdx],
        d_childFitness[totalPairs + pairIdx]
    };
    const int* chrom[4] = {
        d_parentA + base,
        d_parentB + base,
        d_child1 + base,
        d_child2 + base
    };

    int bestIdx = 0, secondIdx = 1;
    if (fits[secondIdx] > fits[bestIdx]) {
        int tmp = bestIdx; bestIdx = secondIdx; secondIdx = tmp;
    }
    for (int i = 2; i < 4; i++) {
        if (fits[i] > fits[bestIdx]) {
            secondIdx = bestIdx;
            bestIdx = i;
        } else if (fits[i] > fits[secondIdx]) {
            secondIdx = i;
        }
    }
    int popOffsetA = pairIdx * 2 * numCities;
    int popOffsetB = popOffsetA + numCities;
    for (int i = 0; i < numCities; i++) {
        d_population[popOffsetA + i] = chrom[bestIdx][i];
        d_population[popOffsetB + i] = chrom[secondIdx][i];
    }
    d_populationFitness[2 * pairIdx] = fits[bestIdx];
    d_populationFitness[2 * pairIdx + 1] = fits[secondIdx];
}

// ====================== CUDA Version Functions with Timing ======================
//
// For the CUDA version, computeTime records the time from the beginning of the function
// until the GPU results are mapped back to the CPU (including CPU->GPU transfers,
// kernel launch and execution). The kernelTime is measured using CUDA events.
// The totalTime is the total time from the start of the function to the end,
// including the mapping of GPU data back to the CPU.

// selectionCUDA: Calls the CPU version of selection (pure CPU phase, kernelTime = 0)
void selectionCUDA(TSP &tsp) {
    auto total_start = high_resolution_clock::now();
    auto compute_start = high_resolution_clock::now();
    selectionCPU(tsp);  // Calls CPU version
    auto compute_end = high_resolution_clock::now();
    double compTime = duration_cast<duration<double>>(compute_end - compute_start).count();
    // No GPU data mapping needed here, so totalTime equals compTime.
    auto total_end = high_resolution_clock::now();
    double totTime = duration_cast<duration<double>>(total_end - total_start).count();
    tsp.selectionTime.computeTime += compTime;
    tsp.selectionTime.kernelTime += 0;
    tsp.selectionTime.totalTime += totTime;
}

// crossoverCUDA: Includes CPU->GPU transfer, kernel execution, and mapping results back to CPU.
// The CPU mapping portion is not included in computeTime.
void crossoverCUDA(TSP &tsp) {
    auto total_start = high_resolution_clock::now();
    auto compute_start = high_resolution_clock::now();
    // 1. Transfer latest flattened parent data from CPU to GPU (included in computeTime)
    hipMemcpy(tsp.d_parentA, tsp.parentAFlat.data(), tsp.parentAFlat.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tsp.d_parentB, tsp.parentBFlat.data(), tsp.parentBFlat.size() * sizeof(int), hipMemcpyHostToDevice);
    int totalPairs = 0;
    for (int i = 0; i < tsp.numIslands; i++) {
        totalPairs += tsp.parentPairs[i].size();
    }
    if (totalPairs == 0) {
        std::cerr << "Warning: No parent pairs available for crossover." << std::endl;
        tsp.offsprings.clear();
        tsp.offsprings.resize(tsp.numIslands);
        auto compute_end = high_resolution_clock::now();
        double compTime = duration_cast<duration<double>>(compute_end - compute_start).count();
        auto total_end = high_resolution_clock::now();
        double totTime = duration_cast<duration<double>>(total_end - total_start).count();
        tsp.crossoverTime.computeTime += compTime;
        tsp.crossoverTime.kernelTime += 0;
        tsp.crossoverTime.totalTime += totTime;
        return;
    }
    int threads = 256;
    int blocks = (totalPairs + threads - 1) / threads;
    unsigned long seed = time(nullptr);

    // Measure kernel execution time using CUDA events
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);
    crossoverKernel<<<blocks, threads>>>(tsp.d_parentA, tsp.d_parentB, tsp.d_child1, tsp.d_child2,
                                           totalPairs, tsp.numCities, tsp.crossoverProbability, seed);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float kernelElapsed = 0;
    hipEventElapsedTime(&kernelElapsed, startEvent, stopEvent); // in milliseconds
    double kTime = kernelElapsed / 1000.0;
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipDeviceSynchronize();
    // 2. Transfer results from GPU to CPU (this mapping is not included in computeTime)
    tsp.offspringFlat.resize(totalPairs * 2 * tsp.numCities);
    hipMemcpy(tsp.offspringFlat.data(), tsp.d_child1, totalPairs * tsp.numCities * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(tsp.offspringFlat.data() + totalPairs * tsp.numCities, tsp.d_child2,
               totalPairs * tsp.numCities * sizeof(int), hipMemcpyDeviceToHost);
    auto compute_end = high_resolution_clock::now();
    double compTime = duration_cast<duration<double>>(compute_end - compute_start).count();
    // 3. CPU mapping phase: reconstruct offspring structure (not included in computeTime)
    auto mapping_start = high_resolution_clock::now();
    Offspring offsprings;
    offsprings.resize(tsp.numIslands);
    int pairIndex = 0;
    for (int island = 0; island < tsp.numIslands; island++) {
        int numPairs = tsp.parentPairs[island].size();
        for (int i = 0; i < numPairs; i++) {
            Individual child1, child2;
            child1.chromosome.resize(tsp.numCities);
            child2.chromosome.resize(tsp.numCities);
            for (int j = 0; j < tsp.numCities; j++) {
                child1.chromosome[j] = tsp.offspringFlat[pairIndex * tsp.numCities + j];
                child2.chromosome[j] = tsp.offspringFlat[(totalPairs + pairIndex) * tsp.numCities + j];
            }
            child1.fitness = 0.0f;
            child2.fitness = 0.0f;
            child1.islandID = island;
            child2.islandID = island;
            offsprings[island].push_back(child1);
            offsprings[island].push_back(child2);
            pairIndex++;
        }
    }
    auto mapping_end = high_resolution_clock::now();
    double mappingTime = duration_cast<duration<double>>(mapping_end - mapping_start).count();
    auto total_end = high_resolution_clock::now();
    double totTime = duration_cast<duration<double>>(total_end - total_start).count();
    // computeTime includes CPU->GPU, kernel execution, and GPU->CPU transfer (but not the mapping phase)
    tsp.crossoverTime.computeTime += compTime;
    tsp.crossoverTime.kernelTime += kTime;
    tsp.crossoverTime.totalTime += totTime;
    tsp.offsprings = offsprings;
}

// mutationCUDA: Includes kernel execution and CPU update of offspring structure.
// The CPU mapping phase is not included in computeTime.
void mutationCUDA(TSP &tsp) {
    auto total_start = high_resolution_clock::now();
    auto compute_start = high_resolution_clock::now();
    int totalPairs = 0;
    for (int i = 0; i < tsp.numIslands; i++) {
        totalPairs += tsp.parentPairs[i].size();
    }
    int totalOffspring = totalPairs * 2;
    int totalGenes = tsp.offspringFlat.size(); // = totalOffspring * tsp.numCities

    // Transfer offspring data from CPU to GPU (included in computeTime)
    hipMemcpy(tsp.d_offspring, tsp.offspringFlat.data(), totalGenes * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (totalOffspring + threads - 1) / threads;
    unsigned long seed = time(nullptr);

    // Record kernel execution time
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);
    mutationKernel<<<blocks, threads>>>(tsp.d_offspring, totalOffspring, tsp.numCities, tsp.mutationProbability, seed);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float kernelElapsed = 0;
    hipEventElapsedTime(&kernelElapsed, startEvent, stopEvent);
    double kTime = kernelElapsed / 1000.0;
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipDeviceSynchronize();

    // Transfer updated offspring data from GPU to CPU (included in computeTime)
    hipMemcpy(tsp.offspringFlat.data(), tsp.d_offspring, totalGenes * sizeof(int), hipMemcpyDeviceToHost);
    auto compute_end = high_resolution_clock::now();
    double compTime = duration_cast<duration<double>>(compute_end - compute_start).count();

    // CPU mapping phase: update offspring structure (not included in computeTime)
    auto mapping_start = high_resolution_clock::now();
    int offset = 0;
    for (int island = 0; island < tsp.numIslands; island++) {
        for (auto &child : tsp.offsprings[island]) {
            for (int j = 0; j < tsp.numCities; j++) {
                child.chromosome[j] = tsp.offspringFlat[offset * tsp.numCities + j];
            }
            offset++;
        }
    }
    auto mapping_end = high_resolution_clock::now();
    double mappingTime = duration_cast<duration<double>>(mapping_end - mapping_start).count();
    auto total_end = high_resolution_clock::now();
    double totTime = duration_cast<duration<double>>(total_end - total_start).count();
    tsp.mutationTime.computeTime += compTime;
    tsp.mutationTime.kernelTime += kTime;
    tsp.mutationTime.totalTime += totTime;
}

// updateOffspringFitnessCUDA: Includes kernel execution and CPU update of offspring fitness.
// The CPU mapping phase (updating the offspring structure) is not included in computeTime.
void updateOffspringFitnessCUDA(TSP &tsp) {
    auto total_start = high_resolution_clock::now();
    auto compute_start = high_resolution_clock::now();
    int totalPairs = 0;
    for (int i = 0; i < tsp.numIslands; i++) {
        totalPairs += tsp.parentPairs[i].size();
    }
    int totalOffspring = totalPairs * 2;
    int totalGenes = tsp.offspringFlat.size();

    // Transfer offspring data from CPU to GPU (included in computeTime)
    hipMemcpy(tsp.d_offspring, tsp.offspringFlat.data(), totalGenes * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (totalOffspring + threads - 1) / threads;

    // Record kernel execution time
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);
    computeFitnessKernel<<<blocks, threads>>>(tsp.d_offspring, tsp.d_distanceMatrix, tsp.d_offspringFitness, tsp.numCities, totalOffspring);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float kernelElapsed = 0;
    hipEventElapsedTime(&kernelElapsed, startEvent, stopEvent);
    double kTime = kernelElapsed / 1000.0;
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipDeviceSynchronize();

    // Transfer computed fitness from GPU to CPU (included in computeTime)
    tsp.offspringFitnessFlat.resize(totalOffspring);
    hipMemcpy(tsp.offspringFitnessFlat.data(), tsp.d_offspringFitness, totalOffspring * sizeof(float), hipMemcpyDeviceToHost);
    auto compute_end = high_resolution_clock::now();
    double compTime = duration_cast<duration<double>>(compute_end - compute_start).count();

    // CPU mapping phase: update the offspring structure (not included in computeTime)
    auto mapping_start = high_resolution_clock::now();
    int idx = 0;
    for (int island = 0; island < tsp.numIslands; island++) {
        for (auto &child : tsp.offsprings[island]) {
            child.fitness = tsp.offspringFitnessFlat[idx++];
        }
    }
    auto mapping_end = high_resolution_clock::now();
    double mappingTime = duration_cast<duration<double>>(mapping_end - mapping_start).count();
    auto total_end = high_resolution_clock::now();
    double totTime = duration_cast<duration<double>>(total_end - total_start).count();
    tsp.updateOffspringFitnessTime.computeTime += compTime;
    tsp.updateOffspringFitnessTime.kernelTime += kTime;
    tsp.updateOffspringFitnessTime.totalTime += totTime;
}

// updatePopulationFitnessCUDA: Includes kernel execution and CPU update of population fitness.
// The CPU mapping phase (updating the population structure) is not included in computeTime.
void updatePopulationFitnessCUDA(TSP &tsp) {
    auto total_start = high_resolution_clock::now();
    // Transfer population data from CPU to GPU
    hipMemcpy(tsp.d_population, tsp.populationFlat.data(), tsp.populationFlat.size() * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (tsp.popSize + threads - 1) / threads;

    // Record kernel execution time
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);
    computeFitnessKernel<<<blocks, threads>>>(tsp.d_population, tsp.d_distanceMatrix, tsp.d_populationFitness, tsp.numCities, tsp.popSize);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float kernelElapsed = 0;
    hipEventElapsedTime(&kernelElapsed, startEvent, stopEvent);
    double kTime = kernelElapsed / 1000.0;
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipDeviceSynchronize();

    std::vector<float> h_fit(tsp.popSize);
    hipMemcpy(h_fit.data(), tsp.d_populationFitness, tsp.popSize * sizeof(float), hipMemcpyDeviceToHost);
    auto compute_end = high_resolution_clock::now();
    double compTime = duration_cast<duration<double>>(compute_end - total_start).count(); // Includes CPU->GPU, kernel, and GPU->CPU transfers
    // CPU mapping phase: update population structure (not included in computeTime)
    auto mapping_start = high_resolution_clock::now();
    int idx = 0;
    for (int island = 0; island < tsp.numIslands; island++) {
        for (auto &ind : tsp.population[island]) {
            ind.fitness = h_fit[idx++];
        }
    }
    auto mapping_end = high_resolution_clock::now();
    double mappingTime = duration_cast<duration<double>>(mapping_end - mapping_start).count();
    auto total_end = high_resolution_clock::now();
    double totTime = duration_cast<duration<double>>(total_end - total_start).count();
    tsp.updatePopulationFitnessTime.computeTime += compTime; // Not subtracting mapping time
    tsp.updatePopulationFitnessTime.kernelTime += kTime;
    tsp.updatePopulationFitnessTime.totalTime += totTime;
}

// replacementCUDA: Includes kernel execution and CPU update of population structure.
// The CPU mapping phase (updating the population structure) is not included in computeTime.
void replacementCUDA(TSP &tsp) {
    auto total_start = high_resolution_clock::now();
    int totalPairs = 0;
    for (int i = 0; i < tsp.numIslands; i++) {
        totalPairs += tsp.parentPairs[i].size();
    }
    // Transfer data from CPU to GPU
    hipMemcpy(tsp.d_parentA, tsp.parentAFlat.data(), tsp.parentAFlat.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tsp.d_parentB, tsp.parentBFlat.data(), tsp.parentBFlat.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tsp.d_parentFitness, tsp.parentFitnessFlat.data(), tsp.parentFitnessFlat.size() * sizeof(float), hipMemcpyHostToDevice);
    int totalGenesPerChild = totalPairs * tsp.numCities;
    hipMemcpy(tsp.d_child1, tsp.offspringFlat.data(), totalGenesPerChild * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tsp.d_child2, tsp.offspringFlat.data() + totalGenesPerChild, totalGenesPerChild * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tsp.d_offspringFitness, tsp.offspringFitnessFlat.data(), tsp.offspringFitnessFlat.size() * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (totalPairs + threads - 1) / threads;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);
    replacementKernel<<<blocks, threads>>>(tsp.d_population, tsp.d_populationFitness,
                                             tsp.d_parentA, tsp.d_parentB, tsp.d_parentFitness,
                                             tsp.d_child1, tsp.d_child2, tsp.d_offspringFitness,
                                             totalPairs, tsp.numCities, totalPairs);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float kernelElapsed = 0;
    hipEventElapsedTime(&kernelElapsed, startEvent, stopEvent);
    double kTime = kernelElapsed / 1000.0;
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipDeviceSynchronize();

    // Transfer GPU results back to CPU (included in computeTime)
    hipMemcpy(tsp.populationFlat.data(), tsp.d_population, tsp.populationFlat.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(tsp.parentFitnessFlat.data(), tsp.d_populationFitness, tsp.parentFitnessFlat.size() * sizeof(float), hipMemcpyDeviceToHost);
    auto compute_end = high_resolution_clock::now();
    double compTime = duration_cast<duration<double>>(compute_end - total_start).count();
    // CPU mapping phase: update population structure (not included in computeTime)
    auto mapping_start = high_resolution_clock::now();
    int offset = 0;
    int fit_idx = 0;
    for (int island = 0; island < tsp.numIslands; island++) {
        for (auto &ind : tsp.population[island]) {
            for (int j = 0; j < tsp.numCities; j++) {
                ind.chromosome[j] = tsp.populationFlat[offset + j];
            }
            ind.fitness = tsp.parentFitnessFlat[fit_idx++];
            offset += tsp.numCities;
        }
    }
    auto mapping_end = high_resolution_clock::now();
    double mappingTime = duration_cast<duration<double>>(mapping_end - mapping_start).count();
    auto total_end = high_resolution_clock::now();
    double totTime = duration_cast<duration<double>>(total_end - total_start).count();
    tsp.replacementTime.computeTime += compTime;
    tsp.replacementTime.kernelTime += kTime;
    tsp.replacementTime.totalTime += totTime;
}

// migrationCUDA: Calls the CPU version of migration and records total time (all included)
void migrationCUDA(TSP &tsp) {
    auto total_start = high_resolution_clock::now();
    auto compute_start = high_resolution_clock::now();
    migrationCPU(tsp);
    auto compute_end = high_resolution_clock::now();
    double compTime = duration_cast<duration<double>>(compute_end - compute_start).count();
    auto mapping_start = high_resolution_clock::now();
    tsp.flattenPopulationToHost();
    auto mapping_end = high_resolution_clock::now();
    double mappingTime = duration_cast<duration<double>>(mapping_end - mapping_start).count();
    auto total_end = high_resolution_clock::now();
    double totTime = duration_cast<duration<double>>(total_end - total_start).count();
    tsp.migrationTime.computeTime += compTime;
    tsp.migrationTime.kernelTime += 0;
    tsp.migrationTime.totalTime += totTime;
}

} // namespace GA
